#include <iostream>
#include <vector>
#include <fstream>
#include <stdexcept>
#include <hip/hip_runtime.h>

#include <sys/stat.h>

// CUDA kernel for tinting, temperature, and saturation adjustments
__global__ void polaroidKernel(unsigned char* img, int width, int height, float3 tint, float temperature, float saturation) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        int idx = (y * width + x) * 3; // RGB has 3 channels

        // Apply temperature adjustment
        float tempRed = img[idx] + temperature;
        float tempGreen = img[idx + 1];
        float tempBlue = img[idx + 2] - temperature;

        // Apply saturation adjustment
        float gray = 0.2989f * tempRed + 0.5870f * tempGreen + 0.1140f * tempBlue;
        float newRed = gray + (tempRed - gray) * saturation;
        float newGreen = gray + (tempGreen - gray) * saturation;
        float newBlue = gray + (tempBlue - gray) * saturation;

        // Apply tint
        newRed = newRed * tint.x;
        newGreen = newGreen * tint.y;
        newBlue = newBlue * tint.z;

        // Clip values to the valid range [0, 255] or use nearest valid value
        newRed = min(255.0f, max(0.0f, newRed));
        newGreen = min(255.0f, max(0.0f, newGreen));
        newBlue = min(255.0f, max(0.0f, newBlue));

        // Apply vignette effect
        float dx = (2.0f * x / width - 1.0f);
        float dy = (2.0f * y / height - 1.0f);
        float dist = sqrtf(dx * dx + dy * dy);
        float vignette = 1.0f - dist * 0.5f;

        // Ensure colors are not altered excessively in dark areas
        float lum = 0.2989f * img[idx] + 0.5870f * img[idx + 1] + 0.1140f * img[idx + 2];
        float lumFactor = lum / 255.0f;
        newRed = min(255.0f, newRed * (0.5f + 0.5f * lumFactor));
        newGreen = min(255.0f, newGreen * (0.5f + 0.5f * lumFactor));
        newBlue = min(255.0f, newBlue * (0.5f + 0.5f * lumFactor));

        img[idx] = static_cast<unsigned char>(newRed * vignette);
        img[idx + 1] = static_cast<unsigned char>(newGreen * vignette);
        img[idx + 2] = static_cast<unsigned char>(newBlue * vignette);
    }
}


void checkCudaErrors(hipError_t err) {
    if (err != hipSuccess) {
        throw std::runtime_error(hipGetErrorString(err));
    }
}

void loadPPM(const std::string& filename, unsigned char*& data, int& width, int& height) {
    std::ifstream file(filename, std::ios::binary);
    if (!file.is_open()) {
        throw std::runtime_error("Unable to open file " + filename);
    }

    std::string header;
    file >> header;
    if (header != "P6") {
        throw std::runtime_error("Invalid PPM file: " + filename);
    }

    file >> width >> height;
    int maxVal;
    file >> maxVal;
    file.ignore(); // Skip the newline character after maxVal

    int dataSize = width * height * 3;
    data = new unsigned char[dataSize];
    file.read(reinterpret_cast<char*>(data), dataSize);
    file.close();
}

void savePPM(const std::string& filename, unsigned char* data, int width, int height) {
    std::ofstream file(filename, std::ios::binary);
    if (!file.is_open()) {
        throw std::runtime_error("Unable to open file " + filename);
    }

    file << "P6\n" << width << " " << height << "\n255\n";
    file.write(reinterpret_cast<char*>(data), width * height * 3);

    file.close();
}

int main(int argc, char* argv[]) {
    printf("%s Starting...\n\n", argv[0]);

    try {
        std::vector<std::string> inputFiles = {"images/img1.ppm", "images/img2.ppm", "images/img3.ppm", "images/img4.ppm", "images/img5.ppm","images/img6.ppm","images/img7.ppm","images/img8.ppm"};

        std::string outputDir = "processed_images";
        // Create the output directory
        #if defined(_WIN32)
            _mkdir(outputDir.c_str());
        #else 
            mkdir(outputDir.c_str(), 0755);
        #endif

        for (const auto& sFilename : inputFiles) {
            unsigned char* h_rgb = nullptr;
            int width, height;
            loadPPM(sFilename, h_rgb, width, height);

            unsigned char* d_rgb;
            checkCudaErrors(hipMalloc(&d_rgb, width * height * 3 * sizeof(unsigned char)));
            checkCudaErrors(hipMemcpy(d_rgb, h_rgb, width * height * 3 * sizeof(unsigned char), hipMemcpyHostToDevice));

            dim3 blockSize(16, 16);
            dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);

            // Applying a Polaroid effect with a slight yellow tint, temperature increase, and saturation adjustment
            polaroidKernel<<<gridSize, blockSize>>>(d_rgb, width, height, make_float3(1.2f, 1.1f, 0.9f), 10.0f, 1.3f);
            hipDeviceSynchronize();

            checkCudaErrors(hipMemcpy(h_rgb, d_rgb, width * height * 3 * sizeof(unsigned char), hipMemcpyDeviceToHost));
            hipFree(d_rgb);

            std::string outputFilename = outputDir + "/" + sFilename.substr(7, sFilename.find_last_of('.') - 7) + "_polaroid.ppm";
            savePPM(outputFilename, h_rgb, width, height);
            std::cout << "Saved image: " << outputFilename << std::endl;

            delete[] h_rgb;
        }

        exit(EXIT_SUCCESS);
    } catch (std::exception& e) {
        std::cerr << "Program error! The following exception occurred: \n" << e.what() << std::endl;
        exit(EXIT_FAILURE);
    } catch (...) {
        std::cerr << "Program error! An unknown type of exception occurred. \n";
        exit(EXIT_FAILURE);
    }

    return 0;
}
